#include "hip/hip_runtime.h"
#include "../headers/export.h"
#include "../headers/hofs.h"
#include "../funcptrs/builtin_tabulate_and_map_intxint.h"
#include "../funcptrs/user_tabulate_int_tuple.h"
#include "../funcptrs/user_map_int_tuple.h"
#include "../funcptrs/builtin_reduce_and_scan_int_tuple.h"
#include "../funcptrs/user_reduce_int_tuple.h"
#include "../funcptrs/user_scan_int_tuple.h"
#include "../funcptrs/builtin_filter_int_tuple.h"
#include "../funcptrs/user_filter_int_tuple.h"
#include "../funcptrs/user_zipwith_int_tuple.h"
#include <stdio.h>
#include <time.h>

#define blockSize = 256

#define threads_reduce 1024
#define block_red_size_reduce (threads_reduce / 32)

#define threads_scan 1024
#define block_red_size_scan (threads_scan / 32)

#define threads_filter 256

//Tabulate
__global__ 
void tabulate_int_tuple_kernel(int* arr_1, int* arr_2, int len, tabulate_fun_int_tuple f){
  
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if(idx >= len){
    return;
  }

  arr_1[idx] = f(idx);
  arr_2[idx] = f(idx);
}

extern "C"
void* tabulate_int_tuple(int size, void* f, Pointer dev_ptr_1, Pointer dev_ptr_1){
  
  tabulate_fun_int_tuple hof = (tabulate_fun_int_tuple)f;
  
  hipMalloc(&(void*)dev_ptr_1, sizeof(int) * size);
  hipMalloc(&(void*)dev_ptr_2, sizeof(int) * size);

  int blockNum = (size / 256) + 1;
  tabulate_int_tuple_kernel<<<blockNum, 256>>>((int*)dev_ptr_1, (int*)dev_ptr_2, size, hof);
  hipDeviceSynchronize();

}

//Map
__global__
void map_int_tuple_kernel(int* arr_1, int* arr_2 int len, map_fun_int_tuple f){
  
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if(idx >= len){
    return;
  }

  std::pair<int, int> tuple = f(arr_1[idx],arr_2[idx]);
  arr_1[idx] = tuple.first
  arr_2[idx] = tuple.second
}
extern "C"
std::pair<void*, void*> map_int_tuple(void* inarr_1, void* inarr_2, void* f, int size){
  
  map_fun_int_tuple hof = (map_fun_int_tuple)f;      
  int blockNum = (size / 256) + 1;
  
  map_intxint_kernel<<<blockNum, 256>>>((int*)inarr_1, (int*)inarr_2, size, hof);

  std::pair<int, int> ret = pair(inarr_1, inarr_2)

  return std::make_pair(inarr_1, inarr_2);

}


__inline__ __device__
std::pair<int,int> warp_red_int_tuple(int t_1, int t_2, reduce_fun_int_tuple f){
  int res_1 = t_1;
  int res_2 = t_2;

  #pragma unroll
  for(int i = warpSize / 2;i > 0;i /= 2){
    int a = __shfl_down(res_1, res_2, i);
    res = f(res_1, res_2, a);
    res_1 = res.first();
    res_2 = res.second();

    //res += a;
  }

  return res;
}

__inline__ __device__
std::pair<int,int> reduce_block_int_tuple(int t_1, int t_2, int b_1, int b_2, reduce_fun_int_tuple f){
  
  // assuming warp size is 32
  // can fix later in the kernel call
  __shared__ int warp_reds_1[block_red_size_reduce];
  __shared__ int warp_reds_2[block_red_size_reduce];

  int warpIdx = threadIdx.x / warpSize;

  int localIdx = threadIdx.x % warpSize;

  std::pair<int,int> inter_res = warp_red_int_tuple(t_1, t_2, f);
  
  if(localIdx == 0){
    warp_reds_1[warpIdx] = inter_res.first;
    warp_reds_2[warpIdx] = inter_res.second;
  }

  __syncthreads();
  
  int broadval2_1 = (threadIdx.x < block_red_size_reduce) ? warp_reds_1[localIdx] : b_1;
  int broadval2_2 = (threadIdx.x < block_red_size_reduce) ? warp_reds_2[localIdx] : b_2;

  std::pair<int, int> res = std::make_pair(b_1, b_2);
  if(warpIdx == 0){
    res = warp_red_int_tuple(broadval2_1, broadval2_2, f);
  }

  return res;
}

__global__
void reduce_int_tuple_kernel(int* in_1, int* in_2, int* out_1, int* our_2, int size, int b_1, int b_2, reduce_fun_int_tuple f){

  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  int sum_1 = b_1;
  int sum_2 = b_2;
  std::pair<int, int> sum;
  
  #pragma unroll
  for(int i = idx; i < size; i += blockDim.x * gridDim.x){
    sum = f(sum_1, sum_2, in_1[i], in_2[i]);
    sum_1 = sum.first;
    sum_2 = sum.second;
    //sum += in[i];
  }
  
  sum = reduce_block_int_tuple(sum_1, sum_2, b_1, b_2, f);
  
  if(threadIdx.x == 0){
    out_1[blockIdx.x] = sum.first;
    out_2[blockIdx.x] = sum.second;
  }
  
}

// cite : https://devblogs.nvidia.com/parallelforall/faster-parallel-reductions-kepler
// for algorithm / ideas on how to use shfl methods for fast reductions
extern "C"
std::pair<int,int> reduce_int_tuple_shfl(void* arr_1, void* arr_2, int size, int b_1, int b_2, void* f){

  reduce_fun_int_tuple hof = (reduce_fun_int_tuple) f;
  

  int numBlocks = (size / threads_reduce) + 1;
  void* res_1;
  void* res_2;

  hipMalloc(&res_1, sizeof(int) * numBlocks);
  hipMalloc(&res_2, sizeof(int) * numBlocks);
  reduce_int_tuple_kernel<<<numBlocks, threads_reduce>>>((int*)arr_1, (int*)arr_2, (int*)res_1, (int*)res_2, 
                                                   size, b_1, b_2, hof);
  reduce_int_tuple_kernel<<<1, 1024>>>((int*)res_1, (int*)res_2, (int*)res_1, (int*)res_2, numBlocks, b_1, b_2, hof);

  std::pair<int, int> ret;
  hipMemcpy(&ret.first,  res_1, sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(&ret.second, res_2, sizeof(int), hipMemcpyDeviceToHost);
  hipFree(res_1);
  cudafree(res_2);

  return ret;
}

//BEGIN SCAN

__device__ __inline__
std::pair<int, int> warp_scan_shfl(int b_1, int b_2, scan_fun_int_tuple f, int* out_1, int* out_2, int idx, int length){
  int warpIdx = threadIdx.x % warpSize;
  std::pair<int, int> res;
  if(idx < length){
    res.first = out_1[idx];
    res.second = out_2[idx];
  }
  else{
    res.first = b_1;
    res.second = b_2;
  }
  #pragma unroll
  for(int i = 1;i < warpSize;i *= 2){
    int a_1 = __shfl_up(res.first, i);
    int a_2 = __shfl_up(res.second, i);
    if(i <= warpIdx){
      res = f(a_1, a_2, res.first, res.second);
    }
  }
  if(idx < length){
    out_1[idx] = res.first;
    out_2[idx] = res.second;
  }
  return res;
}

__device__ __inline__
std::pair<int, int> block_scan(int* in_1, int* in_2, int length, scan_fun_int_tuple f, int b_1, int b_2){

  int idx = threadIdx.x + blockIdx.x * blockDim.x;

  __shared__ int warp_reds[block_red_size_scan];

  int warpIdx = threadIdx.x / warpSize;

  int localIdx= threadIdx.x % warpSize;

  int inter_res = warp_scan_shfl(b, f, in, idx, length);

  if(localIdx == warpSize - 1){
    warp_reds[warpIdx] = inter_res;
  }

  __syncthreads();

  int res = b;
  if(warpIdx == 0){
    res = warp_scan_shfl(b, f, warp_reds, localIdx, block_red_size_scan);
  }
  
  __syncthreads();

  if(idx < length && warpIdx != 0){
    in[idx] = f(warp_reds[warpIdx - 1], in[idx]);
  }

  //warp number 0, lane number block_red_size_scan 
  //will return the final result for scanning over this
  //block 
  return res;
}

//inclusive kernel
__global__
void scan_intxint_kernel(int* in, int* block_results, scan_fun_intxint f, int b, int length){
  
  int block_res = block_scan(in, length, f, b);
  if(threadIdx.x == block_red_size_scan - 1){
    block_results[blockIdx.x] = block_res;
  }
}
__global__
void compress_results(int* block_res, int* out, int len, scan_fun_intxint f){
  int idx = threadIdx.x + blockIdx.x * blockDim.x;

  if(blockIdx.x == 0){
    return;
  }
  else{
    if(idx < len){
      out[idx] = f(block_res[blockIdx.x - 1], out[idx]);
    }
  }
}

//this is terrible
__global__
void serial_scan(int* bres, int len, int b, scan_fun_intxint f){
  int res = b;
  #pragma unroll
  for(int i = 0;i < len;i++){
    res = f(res, bres[i]);
    bres[i] = res;
  }
}

extern "C"
void* inclusive_scan_intxint(void* in, void* f, int length, int b){
  
  scan_fun_intxint hof = (scan_fun_intxint)f;

  int num_blocks_first = (length / threads_scan) + 1;
  int* block_results;
  int* dummy;
  hipMalloc(&block_results, sizeof(int) * num_blocks_first);
  hipMalloc(&dummy, sizeof(int));

  scan_intxint_kernel<<<num_blocks_first, threads_scan>>>
          ((int*)in, block_results, hof, b, length);

  if(num_blocks_first == 1){
    hipDeviceSynchronize();
    hipFree(block_results);
    hipFree(dummy);
    return in;
  }
  else if(num_blocks_first <= 1024){
    scan_intxint_kernel<<<1, 1024>>>(block_results, dummy, hof, b, num_blocks_first);
    compress_results<<<num_blocks_first, threads_scan>>>(block_results, (int*)in, length, hof);
    hipDeviceSynchronize();
    hipFree(block_results);
    hipFree(dummy);
    return in;
  }
  else{
    int leftover = (num_blocks_first / threads_scan) + 1;
    int* block_block_results;
    hipMalloc(&block_block_results, sizeof(int) * leftover);
    scan_intxint_kernel<<<leftover, threads_scan>>>
            (block_results, block_block_results, hof, b, num_blocks_first);
    serial_scan<<<1,1>>>(block_block_results, leftover, b, hof);
    compress_results<<<leftover, threads_scan>>>
            (block_block_results, block_results, num_blocks_first, hof);
    compress_results<<<num_blocks_first, threads_scan>>>(block_results, (int*)in, length, hof);
    hipDeviceSynchronize();
    hipFree(block_results);
    hipFree(dummy);
    hipFree(block_block_results);
    return in;
  }
}

//BEGIN EXCLUSIVE SCAN

__global__
void excl_compress_results(int* block_res, int* out, int len, scan_fun_intxint f, int* final, int b){
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if(idx >= len) return;
  if(blockIdx.x != 0){
    out[idx] = f(block_res[blockIdx.x - 1], out[idx]);
  }
  __syncthreads();
  int toWrite = b;
  if(threadIdx.x == 0){
    if(idx == 0){
      toWrite = b;
    }
    else{
      toWrite = block_res[blockIdx.x - 1];
    }
  }
  else{
    toWrite = out[idx - 1];
  }
  if(idx == len - 1){
    *final = out[idx];
  }
  __syncthreads();
  out[idx] = toWrite;
}

extern "C"
int exclusive_scan_intxint(void* in, void* f, int length, int b){
  
  scan_fun_intxint hof = (scan_fun_intxint)f;

  int num_blocks_first = (length / threads_scan) + 1;
  int* block_results;
  int* dummy;
  int* final_val;
  hipMalloc(&block_results, sizeof(int) * num_blocks_first);
  hipMalloc(&dummy, sizeof(int));
  hipMalloc(&final_val, sizeof(int));


  scan_intxint_kernel<<<num_blocks_first, threads_scan>>>
          ((int*)in, block_results, hof, b, length);
  int res;
  if(num_blocks_first == 1){
    excl_compress_results<<<num_blocks_first, threads_scan>>>
          (block_results, (int*)in, length, hof, final_val, b);
    hipMemcpy(&res, final_val, sizeof(int), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    hipFree(block_results);
    hipFree(dummy);
    hipFree(final_val);
    return res;
  }
  else if(num_blocks_first <= 1024){
    scan_intxint_kernel<<<1, 1024>>>(block_results, dummy, hof, b, num_blocks_first);
    excl_compress_results<<<num_blocks_first, threads_scan>>>
            (block_results, (int*)in, length, hof, final_val, b);
    hipMemcpy(&res, final_val, sizeof(int), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    hipFree(block_results);
    hipFree(dummy);
    hipFree(final_val);
    return res;
  }
  else{
    int leftover = (num_blocks_first / threads_scan) + 1;
    int* block_block_results;
    hipMalloc(&block_block_results, sizeof(int) * leftover);
    scan_intxint_kernel<<<leftover, threads_scan>>>
            (block_results, block_block_results, hof, b, num_blocks_first);
    serial_scan<<<1,1>>>(block_block_results, leftover, b, hof);
    compress_results<<<leftover, threads_scan>>>
            (block_block_results, block_results, num_blocks_first, hof);
    excl_compress_results<<<num_blocks_first, threads_scan>>>
            (block_results, (int*)in, length, hof, final_val, b);
    hipMemcpy(&res, final_val, sizeof(int), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    hipFree(block_results);
    hipFree(dummy);
    hipFree(final_val);
    hipFree(block_block_results);
    return res;
  }
}

__global__
void filter_map(int* in, int* out1, int len, filter_fun_intxint f){
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if(idx < len){
    if(f(in[idx])){
      out1[idx] = 1;
    }
    else{
      out1[idx] = 0;
    }
  }
}
__global__
void squish(int* in, int* scanned, int* out, int length, filter_fun_intxint f){
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  
  if(idx < length){
    if(f(in[idx]) == 1){
      out[scanned[idx]] = in[idx];
    }
  }
}

extern "C"
void* filter_intxint(void* arr, int length, void* f, Pointer outlen){
  filter_fun_intxint hof = (filter_fun_intxint)f;
  
  int blocks = (length / threads_filter) + 1;
    
  // make buffer array

  // this map could have been fused in with the scan with some 
  // extra code copy pasta i didnt want to do

  int* scanned;
  hipMalloc(&scanned, sizeof(int) * length);
  filter_map<<<blocks, threads_filter>>>((int*)arr, scanned, length, hof);
  
  //scan over the bits
  reduce_fun_intxint add = (reduce_fun_intxint)gen_add_intxint();
  int len = exclusive_scan_intxint(scanned, (void*)add, length, 0);

  int* res;
  hipMalloc(&res, sizeof(int) * len);

  squish<<<blocks, threads_filter>>>((int*)arr, scanned, res, length, hof);
  *(int*)outlen = len;
  //hipFree(bits);
  hipFree(scanned);
  return res;
}

__global__
void zipsquish(int* arr1, int* arr2, int* out, zipwith_fun_intxint f, int length){
  int idx = threadIdx.x + blockIdx.x * blockDim.x;

  if(idx < length){
    out[idx] = f(arr1[idx], arr2[idx]);
  }
}

extern "C"
void* zipwith_intxint(void* arr1, void* arr2, void* f, int length){

  zipwith_fun_intxint hof = (zipwith_fun_intxint)f;
  
  int* res;
  hipMalloc(&res, sizeof(int) * length);

  int blocks = (length / threads_filter) + 1;
  zipsquish<<<blocks, threads_filter>>>((int*)arr1, (int*)arr2, res, hof, length);

  hipDeviceSynchronize();
  return res;
}
//Reduce - cite http://developer.download.nvidia.com/compute/cuda/1.1-Beta/x86_website/projects/reduction/doc/reduction.pdf - another reduction algorithm choice
