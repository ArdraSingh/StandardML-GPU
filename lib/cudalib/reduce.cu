#include "hip/hip_runtime.h"
#include "../headers/hofs.h"
#include "../headers/export.h"
#include <stdio.h>

#define threads_reduce 256
#define block_red_size_reduce (threads_reduce / 32)

__inline__ __device__
int warp_red_int(int t, reduce_fun_int f){
  int res = t;
  for(int i = warpSize / 2;i >= 1;i /= 2){
    int a = __shfl_down(res, i);
    res = (*f)(res, a);
  }
  return res;
}

__inline__ __device__
int reduce_block_int(int t, int b, reduce_fun_int f){
  
  // assuming warp size is 32
  // can fix later in the kernel call
  __shared__ int warp_reds[block_red_size_reduce];

  int warpIdx = threadIdx.x / warpSize;

  int localIdx = threadIdx.x % warpSize;

  // need to handle case where length of array is not
  // exactly equal to the block size

  int inter_res = warp_red_int(t, f);
  
  if(localIdx == 0){
    warp_reds[warpIdx] = inter_res;
  }

  __syncthreads();
  
  int broadval2 = (localIdx < block_red_size_reduce) ? warp_reds[localIdx] : b;
  int res = b;
  if(warpIdx == 0){
    res = warp_red_int(broadval2, f);
  }

  return res;
}

__global__
void reduce_int_kernel(int* in, int* out, int size, int b, reduce_fun_int f){

  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if(idx == 0){
    printf("got here\n");
  }
  int sum = b;

  for(int i = idx;i < size;i += blockDim.x * gridDim.x){
    sum = (*f)(b,sum);
  }
  
  //sum = reduce_block_int(sum, b, f);
  /*
  if(threadIdx.x == 0){
    while(true){
      int t = *out;
      int final = (*f)(sum, t);
      int res = atomicCAS(out, t, final);
      if(res != t){
        break;
      }
    }
  }
  */
}


extern "C"
int reduce_int_shfl(void* arr, int size, int b, void* f){
  reduce_fun_int hof = (reduce_fun_int)f;
  
  void* res;
  hipMalloc(&res, sizeof(int));

  int numBlocks = (size / threads_reduce) + 1;
  reduce_int_kernel<<<numBlocks, threads_reduce>>>((int*)arr, (int*)res, 
                                                   size, b, hof);

  hipDeviceSynchronize();
  hipError_t wei = hipGetLastError();
  printf("%s\n", hipGetErrorString(wei));

  int ret;
  hipMemcpy(&ret, res, sizeof(int), hipMemcpyDeviceToHost);
  printf("%d\n", ret);
  return ret;
}
