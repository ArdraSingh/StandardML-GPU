#include "export.h"

extern "C"
void* allocate_on_gpu(size_t size, int smltype){
	size_t typesize;
	if(smltype == 0){
		typesize = sizeof(int);
	}
	else{
		typesize = sizeof(float);
	}

	void* ret_ptr;
	hipMalloc(&ret_ptr, typesize * size);
	hipMemset(ret_ptr, 0, typesize * size);
	return ret_ptr;
}

extern "C"
void* copy_float_into_gpu(Pointer src, int size){
	
	void* ret_ptr;
	hipMalloc(&ret_ptr, sizeof(float) * size);
  hipMemcpy(ret_ptr, src, sizeof(float) * size, hipMemcpyHostToDevice);
  
  return ret_ptr;
}

extern "C"
void* copy_int_into_gpu(Pointer src, int size){
	
	void* ret_ptr;
	hipMalloc(&ret_ptr, sizeof(int) * size);
  hipMemcpy(ret_ptr, src, sizeof(int) * size, hipMemcpyHostToDevice);
  
  return ret_ptr;
}

extern "C"
void copy_float_gpu(Pointer dest, void* gpuarr, size_t size){
	size_t typesize = sizeof(float);
  hipMemcpy(dest, gpuarr, size * typesize, hipMemcpyDeviceToHost);
}

extern "C"
void copy_int_gpu(Pointer dest, void* gpuarr, size_t size){
	size_t typesize = sizeof(int);
  hipMemcpy(dest, gpuarr, size * typesize, hipMemcpyDeviceToHost);
}

extern "C"
void free_gpu_ptr(void* ptr){
	hipFree(ptr);
}
