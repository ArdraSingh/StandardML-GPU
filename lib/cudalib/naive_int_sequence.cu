#include "hip/hip_runtime.h"
#include "../headers/export.h"
#include "../headers/hofs.h"
#include "../funcptrs/builtin_tabulate_and_map_int.h"
#include "../funcptrs/user_tabulate_int.h"
#include "../funcptrs/builtin_map_int.h"
#include "../funcptrs/user_map_int.h"
#include "../funcptrs/builtin_reduce_and_scan_int.h"
#include "../funcptrs/user_reduce_int.h"
#include <stdio.h>
#include <time.h>

#define blockSize = 256

#define threads_reduce 1024
#define block_red_size_reduce (threads_reduce / 32)

#define threads_scan 1024
#define block_red_size_scan (threads_scan / 32)

//Tabulate
__global__ 
void tabulate_int_kernel(int* arr, int len, tabulate_fun_int f){
  
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if(idx >= len){
    return;
  }

  arr[idx] = f(idx);

}
extern "C"
void* tabulate_int(void* f, int size){
  
  tabulate_fun_int hof = (tabulate_fun_int)f;
  
  void* dev_ptr;
  hipMalloc(&dev_ptr, sizeof(int) * size);

  int blockNum = (size / 256) + 1;
  tabulate_int_kernel<<<blockNum, 256>>>((int*)dev_ptr, size, hof);
  
  return dev_ptr;

}

//Map
__global__
void map_int_kernel(int* arr, int len, map_fun_int f){
  
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if(idx >= len){
    return;
  }

  arr[idx] = f(arr[idx]);

}
extern "C"
void* map_int(void* inarr, void* f, int size){
  
  map_fun_int hof = (map_fun_int)f;      
  int blockNum = (size / 256) + 1;
  
  map_int_kernel<<<blockNum, 256>>>((int*)inarr, size, hof);

  return inarr;

}


__inline__ __device__
int warp_red_int(int t, reduce_fun_int f){
  int res = t;

  #pragma unroll
  for(int i = warpSize / 2;i > 0;i /= 2){
    int a = __shfl_down(res, i);
    res = f(res, a);
    //res += a;
  }
  return res;
}

__inline__ __device__
int reduce_block_int(int t, int b, reduce_fun_int f){
  
  // assuming warp size is 32
  // can fix later in the kernel call
  __shared__ int warp_reds[block_red_size_reduce];

  int warpIdx = threadIdx.x / warpSize;

  int localIdx = threadIdx.x % warpSize;

  int inter_res = warp_red_int(t, f);
  
  if(localIdx == 0){
    warp_reds[warpIdx] = inter_res;
  }

  __syncthreads();
  
  int broadval2 = (threadIdx.x < block_red_size_reduce) ? warp_reds[localIdx] : b;
  int res = b;
  if(warpIdx == 0){
    res = warp_red_int(broadval2, f);
  }

  return res;
}

__global__
void reduce_int_kernel(int* in, int* out, int size, int b, reduce_fun_int f){

  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  int sum = b;
  
  #pragma unroll
  for(int i = idx;i < size;i += blockDim.x * gridDim.x){
    sum = f(sum,in[i]);
    //sum += in[i];
  }
  
  sum = reduce_block_int(sum, b, f);
  
  if(threadIdx.x == 0){
    out[blockIdx.x] = sum;
  }
  
}

// cite : https://devblogs.nvidia.com/parallelforall/faster-parallel-reductions-kepler
// for algorithm / ideas on how to use shfl methods for fast reductions
extern "C"
int reduce_int_shfl(void* arr, int size, int b, void* f){

  reduce_fun_int hof = (reduce_fun_int)f;
  

  int numBlocks = (size / threads_reduce) + 1;
  void* res;
  hipMalloc(&res, sizeof(int) * numBlocks);
  reduce_int_kernel<<<numBlocks, threads_reduce>>>((int*)arr, (int*)res, 
                                                   size, b, hof);
  reduce_int_kernel<<<1, 1024>>>((int*)res, (int*)res, numBlocks, b, hof);

  int ret;
  hipMemcpy(&ret, res, sizeof(int), hipMemcpyDeviceToHost);
  hipFree(res);
  return ret;
}

//BEGIN SCAN

__device__ __inline__
int warp_scan_shfl(int b, scan_fun_int f, int* out, int idx, int length){
  int warpIdx = threadIdx.x % warpSize;
  int res;
  if(idx < length){
    res = out[idx];
  }
  else{
    res = b;
  }
  #pragma unroll
  for(int i = 1;i < warpSize;i *= 2){
    int a = __shfl_up(res, i);
    if(i <= warpIdx){
      res = f(a, res);
    }
  }
  if(idx < length){
    out[idx] = res;
  }
  return res;
}

__device__ __inline__
int block_scan(int* in, int length, scan_fun_int f, int b){

  int idx = threadIdx.x + blockIdx.x * blockDim.x;

  __shared__ int warp_reds[block_red_size_scan];

  int warpIdx = threadIdx.x / warpSize;

  int localIdx= threadIdx.x % warpSize;

  int inter_res = warp_scan_shfl(b, f, in, idx, length);

  if(localIdx == warpSize - 1){
    warp_reds[warpIdx] = inter_res;
  }

  __syncthreads();

  int res = b;
  if(warpIdx == 0){
    res = warp_scan_shfl(b, f, warp_reds, localIdx, block_red_size_scan);
  }
  
  __syncthreads();

  if(idx < length && warpIdx != 0){
    in[idx] = f(warp_reds[warpIdx - 1], in[idx]);
  }

  //warp number 0, lane number block_red_size_scan 
  //will return the final result for scanning over this
  //block 
  return res;
}

//inclusive kernel
__global__
void scan_int_kernel(int* in, int* block_results, scan_fun_int f, int b, int length){
  
  int block_res = block_scan(in, length, f, b);
  if(threadIdx.x == block_red_size_scan - 1){
    block_results[blockIdx.x] = block_res;
  }
}
__global__
void compress_results(int* block_res, int* out, int len, scan_fun_int f){
  int idx = threadIdx.x + blockIdx.x * blockDim.x;

  if(blockIdx.x == 0){
    return;
  }
  else{
    if(idx < len){
      out[idx] = f(block_res[blockIdx.x - 1], out[idx]);
    }
  }
}

//this is terrible
__global__
void serial_scan(int* bres, int len, int b, scan_fun_int f){
  int res = b;
  #pragma unroll
  for(int i = 0;i < len;i++){
    res = f(res, bres[i]);
    bres[i] = res;
  }
}

extern "C"
void* inclusive_scan_int(void* in, void* f, int length, int b){
  
  scan_fun_int hof = (scan_fun_int)f;

  int num_blocks_first = (length / threads_scan) + 1;
  int* block_results;
  int* dummy;
  hipMalloc(&block_results, sizeof(int) * num_blocks_first);
  hipMalloc(&dummy, sizeof(int));

  scan_int_kernel<<<num_blocks_first, threads_scan>>>
          ((int*)in, block_results, hof, b, length);

  if(num_blocks_first == 1){
    hipDeviceSynchronize();
    hipFree(block_results);
    hipFree(dummy);
    return in;
  }
  else if(num_blocks_first <= 1024){
    scan_int_kernel<<<1, 1024>>>(block_results, dummy, hof, b, num_blocks_first);
    compress_results<<<num_blocks_first, threads_scan>>>(block_results, (int*)in, length, hof);
    hipDeviceSynchronize();
    hipFree(block_results);
    hipFree(dummy);
    return in;
  }
  else{
    int leftover = (num_blocks_first / threads_scan) + 1;
    int* block_block_results;
    hipMalloc(&block_block_results, sizeof(int) * leftover);
    scan_int_kernel<<<leftover, threads_scan>>>
            (block_results, block_block_results, hof, b, num_blocks_first);
    serial_scan<<<1,1>>>(block_block_results, leftover, b, hof);
    compress_results<<<leftover, threads_scan>>>
            (block_block_results, block_results, num_blocks_first, hof);
    compress_results<<<num_blocks_first, threads_scan>>>(block_results, (int*)in, length, hof);
    hipDeviceSynchronize();
    hipFree(block_results);
    hipFree(dummy);
    hipFree(block_block_results);
    return in;
  }
}

//BEGIN EXCLUSIVE SCAN

__device__ __inline__
int warp_scan_shfl_exl(int b, scan_fun_int f, int* out, int idx, int length){
  int warpIdx = threadIdx.x % warpSize;
  int res;
  if(warpIdx == 0){
    res = b;
  }
  else{
    if(idx - 1 > length){
      res = b;
    }
    else{
      res = out[idx - 1];
    }
  }
  #pragma unroll
  for(int i = 1;i < warpSize;i *= 2){
    int a = __shfl_up(res, i);
    if(i <= warpIdx){
      res = f(a, res);
    }
  }
  if(idx < length){
    out[idx] = res;
  }
  return res;
}

__device__ __inline__
int excl_block_scan(int* in, int length, scan_fun_int f, int b){

  int idx = threadIdx.x + blockIdx.x * blockDim.x;

  __shared__ int warp_reds[block_red_size_scan];

  int warpIdx = threadIdx.x / warpSize;

  int localIdx= threadIdx.x % warpSize;

  int inter_res = warp_scan_shfl(b, f, in, idx, length);

  if(localIdx == warpSize - 1){
    warp_reds[warpIdx] = inter_res;
  }

  __syncthreads();

  int res = b;
  if(warpIdx == 0){
    res = warp_scan_shfl(b, f, warp_reds, localIdx, block_red_size_scan);
  }
  
  __syncthreads();

  if(idx < length && warpIdx != 0){
    in[idx] = f(warp_reds[warpIdx - 1], in[idx]);
  }

  //warp number 0, lane number block_red_size_scan 
  //will return the final result for scanning over this
  //block
  return res;
}

//inclusive kernel
__global__
void excl_scan_int_kernel(int* in, int* block_results, scan_fun_int f, int b, int length, int* out){
  
  int block_res = block_scan(in, length, f, b);
  if(threadIdx.x == block_red_size_scan - 1){
    block_results[blockIdx.x] = block_res;
    *out = block_res;
  }
}
__global__
void excl_compress_results(int* block_res, int* out, int len, scan_fun_int f, int* final, int b){
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if(idx >= len) return;
  if(blockIdx.x != 0){
    out[idx] = f(block_res[blockIdx.x - 1], out[idx]);
  }
  __syncthreads();
  int toWrite = b;
  if(threadIdx.x == 0){
    if(idx == 0){
      toWrite = b;
    }
    else{
      toWrite = block_res[blockIdx.x - 1];
    }
  }
  else{
    toWrite = out[idx - 1];
  }
  if(idx == len - 1){
    *final = out[idx];
  }
  __syncthreads();
  out[idx] = toWrite;
}

extern "C"
int exclusive_scan_int(void* in, void* f, int length, int b){
  
  scan_fun_int hof = (scan_fun_int)f;

  int num_blocks_first = (length / threads_scan) + 1;
  int* block_results;
  int* dummy;
  int* final_val;
  hipMalloc(&block_results, sizeof(int) * num_blocks_first);
  hipMalloc(&dummy, sizeof(int));
  hipMalloc(&final_val, sizeof(int));


  scan_int_kernel<<<num_blocks_first, threads_scan>>>
          ((int*)in, block_results, hof, b, length);
  int res;
  if(num_blocks_first == 1){
    excl_compress_results<<<num_blocks_first, threads_scan>>>
          (block_results, (int*)in, length, hof, final_val, b);
    hipMemcpy(&res, final_val, sizeof(int), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    hipFree(block_results);
    hipFree(dummy);
    hipFree(final_val);
    return res;
  }
  else if(num_blocks_first <= 1024){
    scan_int_kernel<<<1, 1024>>>(block_results, dummy, hof, b, num_blocks_first);
    excl_compress_results<<<num_blocks_first, threads_scan>>>
            (block_results, (int*)in, length, hof, final_val, b);
    hipMemcpy(&res, final_val, sizeof(int), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    hipFree(block_results);
    hipFree(dummy);
    hipFree(final_val);
    return res;
  }
  else{
    int leftover = (num_blocks_first / threads_scan) + 1;
    int* block_block_results;
    hipMalloc(&block_block_results, sizeof(int) * leftover);
    scan_int_kernel<<<leftover, threads_scan>>>
            (block_results, block_block_results, hof, b, num_blocks_first);
    serial_scan<<<1,1>>>(block_block_results, leftover, b, hof);
    compress_results<<<leftover, threads_scan>>>
            (block_block_results, block_results, num_blocks_first, hof);
    excl_compress_results<<<num_blocks_first, threads_scan>>>
            (block_results, (int*)in, length, hof, final_val, b);
    hipMemcpy(&res, final_val, sizeof(int), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    hipFree(block_results);
    hipFree(dummy);
    hipFree(final_val);
    hipFree(block_block_results);
    return res;
  }
}

/*
//Reduce - cite http://developer.download.nvidia.com/compute/cuda/1.1-Beta/x86_website/projects/reduction/doc/reduction.pdf
__global__
void reduce_int_kernel(int* arr, int len, reduce_fun_int f){
  
  extern __shared__ int sdata[];
  unsigned int blockSize = blockDim.x;
  unsigned int thread_id = threadIdx.x;
  unsigned int array_id = blockIdx.x * (blockDim.x * 2) + thread_id;
  unsigned int gridSize = 2 * blockDim.x * gridDim.x;
  sdata[thread_id] = 0;

  while (array_id < len) 
  {
    sdata[thread_id] += arr[i] + arr[array_id+ blockSize];
    array_id += blockSize;
  }

  __syncthreads();

  if(thread_id < 128) 
    sdata[thread_id] += sdata[thread_id + 128];

  __syncthreads();

  if (thread_id <  64)
    sdata[thread_id] += sdata[thread_id + 64];

  __syncthreads();

  if (thread_id < 32)
  {
    sdata[thread_id] += sdata[thread_id + 32];
    sdata[thread_id] += sdata[thread_id + 16];
    sdata[thread_id] += sdata[thread_id + 8];
    sdata[thread_id] += sdata[thread_id + 4];
    sdata[thread_id] += sdata[thread_id + 1];
  }
  
  if (thread_id ==0)
    arr[blockIdx.x] = sdata[0];

}

void* reduce_int(void* inarr, void* f, int size){
  
  reduce_fun_int hof = (reduce_fun_int)f;      
  int blockNum = (size / 256) + 1;
  
  reduce_int_kernel<<<blockNum, 256>>>((int*)inarr, size, hof);

  return inarr;

}
*/
