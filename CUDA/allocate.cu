#include "export.h"

extern "C"
void* allocate_on_gpu(size_t size, int smltype){
	size_t typesize;
	if(smltype == 0){
		typesize = sizeof(int);
	}
	else{
		typesize = sizeof(float);
	}

	void* ret_ptr;
	hipMalloc(&ret_ptr, typesize * size);
	hipMemset(ret_ptr, 0, typesize * size);
	return ret_ptr;
}

extern "C"
void copy_float_gpu(Pointer dest, void* gpuarr, size_t size){
	size_t typesize = sizeof(float);
  hipMemcpy(dest, gpuarr, size * typesize, hipMemcpyDeviceToHost);
}

extern "C"
void copy_int_gpu(Pointer dest, void* gpuarr, size_t size){
	size_t typesize = sizeof(int);
  hipMemcpy(dest, gpuarr, size * typesize, hipMemcpyDeviceToHost);
}

extern "C"
void free_gpu_ptr(void* ptr){
	hipFree(ptr);
}
