#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include <stdio.h>
#include <time.h>

int main(int argc, char** argv){
  int size = atoi(argv[1]);
  
  thrust::device_vector<int> test(size);
  thrust::fill(test.begin(), test.end(), 1);
  
  clock_t begin = clock();
  thrust::inclusive_scan(test.begin(), test.end(), test.begin(), thrust::plus<int>());
  clock_t end = clock();
  printf("time spent : %.4f\n", (double) (end - begin) / CLOCKS_PER_SEC);
  return 0;
}
